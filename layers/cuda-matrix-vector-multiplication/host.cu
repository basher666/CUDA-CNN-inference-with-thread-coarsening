#include "gen_gpu.h"


//Function To handle any errors occurred in the function calls
int funcCheck(hipError_t stmt){
     do {
            hipError_t err = stmt;
            if (err != hipSuccess) {
                printf( "Failed to run stmt %d ", __LINE__);
                return -1;
            }
        } while(0);
    return 0;
}

int numARows;   // number of rows in the matrix A
int numAColumns;  // number of columns in the matrix A
int numBRows;   // number of rows in the matrix B
int numBColumns=1;  // number of columns in the matrix B
int numCRows;  // number of rows in the matrix C (you have to set this)
int numCColumns=1; // number of columns in the matrix C (you have to set this)
int nelem_per_thread; // THread coarsening factor

//*************************************************************
void Print_Mat(int Row,int Col,float *Mat)//Function To print the Matrix
{
 int tot = Row*Col;
 for(int i=0;i<tot;i++)
   {
   printf("%f  ",Mat[i]);

   if((i%Col)==0 )
    {
     printf("\n");
    }
   }
}

//Function close
//*************************************************************
//Normal CPU Matrix Multiplication
void matMultiplyAddOnHost(float * A, float * B, float * C, float *bias, int numARows,
                        int numAColumns, int numBRows, int numBColumns,
                        int numCRows, int numCColumns)
{
    for (int i=0; i < numARows; i ++)
    {
        for (int j = 0; j < numBColumns; j++)
        {
            float c = 0.0;
            for (int k = 0; k < numAColumns; k++)
            {
                c += A[i*numAColumns + k] * B[k*numBColumns + j];
            }
	    C[i*numCColumns + j] = c + bias[i*numCColumns + j];
        }
    }
}
//*************************************************************
int main(int argc, char ** argv) {
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * hostBias; // The Bias Matrix
    float * hostComputedC;
    float * deviceA;
    float * deviceB;
    float * deviceC;
    float * deviceBias;

    // Please adjust rows and columns according to you need.

    // printf("\nPlease Enter Rows and Columns of A:");
    // scanf("%d %d",&numARows,&numAColumns);
    numARows = 1000;
    numAColumns = 500;
    numBRows = 500;
    nelem_per_thread = 4;
    // printf("\nPlease Enter Rows of B:");
    // scanf("%d %d",&numBRows);

    hostA = (float *) malloc(sizeof(float)*numARows*numAColumns);
    hostB = (float *) malloc(sizeof(float)*numBRows*numBColumns);

    for (int i = 0; i < numARows*numAColumns; i++)//Matrix Initialization
    {
        hostA[i]=1.0;
    }
    for (int i = 0; i < numBRows*numBColumns; i++)
    {
        hostB[i]=1.0;
    }

    printf("\nMatrix A Values:\n");
    Print_Mat(numARows,numAColumns,hostA);//Function Call

    printf("\n\nMatrix B Values:\n");
    Print_Mat(numBRows,numBColumns,hostB);//Function Call



    // Setting numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;

    hostC = (float *) malloc(sizeof(float)*numCRows*numCColumns);
    hostBias = (float *) malloc(sizeof(float)*numCRows*numCColumns);
    hostComputedC = (float *) malloc(sizeof(float)*numCRows*numCColumns);

    for (int i = 0; i < numCRows*numCColumns; i++)
    {
        hostBias[i]=1.0;
    }

    // Allocating GPU memory
    funcCheck(hipMalloc((void **)&deviceA, sizeof(float)*numARows*numAColumns));
    funcCheck(hipMalloc((void **)&deviceB, sizeof(float)*numBRows*numBColumns));
    funcCheck(hipMalloc((void **)&deviceC, sizeof(float)*numCRows*numCColumns));
    funcCheck(hipMalloc((void **)&deviceBias, sizeof(float)*numCRows*numCColumns));

    // Copy memory to the GPU
    funcCheck(hipMemcpy(deviceA, hostA, sizeof(float)*numARows*numAColumns, hipMemcpyHostToDevice));
    funcCheck(hipMemcpy(deviceB, hostB, sizeof(float)*numBRows*numBColumns, hipMemcpyHostToDevice));
    funcCheck(hipMemcpy(deviceBias, hostBias, sizeof(float)*numCRows*numCColumns, hipMemcpyHostToDevice));

    // Initialize the grid and block dimensions
    // Launch the Vector Add CUDA Kernel
    int numThreadsReq = (numCRows+nelem_per_thread-1)/nelem_per_thread;
    int threadsPerBlock = 256;
    int blocksPerGrid =(numThreadsReq + threadsPerBlock - 1) / threadsPerBlock;
    dim3 dimGrid(blocksPerGrid, 1, 1);//Number of Blocks required
    dim3 dimBlock(threadsPerBlock, 1, 1);//Number of threads in each block
	
    // Shared memory for parameter vetor and bias values
    int totSharedMem = (numAColumns + numCRows*numCColumns)* sizeof(float); // Shared memory per block
    //int totSharedMem = (threadsPerBlock * nelem_per_thread * numAColumns + numAColumns + numCRows*numCColumns)* sizeof(float); // Shared memory per block

    // float *A, float *x, float *y, const int m, const int n
    //@@ Launch the GPU Kernel here

    printf("CUDA kernel launch with %d blocks of %d threads, and %d of shared Memory\n", blocksPerGrid, threadsPerBlock, totSharedMem);

    gen_matvec<<<dimGrid, dimBlock, totSharedMem>>>(deviceA, deviceB, deviceC, deviceBias, numCRows, numAColumns, nelem_per_thread);

    hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

    //hipDeviceSynchronize();//To synchronize the device

    // Copy the results in GPU memory back to the CPU
    funcCheck(hipMemcpy(hostC, deviceC, sizeof(float)*numCRows*numCColumns, hipMemcpyDeviceToHost));

    printf("\nMatrix C From Device\n");
    Print_Mat(numCRows,numCColumns,hostC);//Function Call

    matMultiplyAddOnHost(hostA, hostB, hostComputedC, hostBias, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    printf("\nMatrix C From Host\n");
    Print_Mat(numCRows,numCColumns,hostComputedC);//Function Call

    for (int i=0; i < numCColumns*numCRows; i++)//Compare both the result matrices 1. MatrixMultiplyonHost 2. MatrixMultiplyonDevice
    {
        if (hostComputedC[i]  != hostC[i] )
        {
            printf("Mismatch at Row = %d Col = %d hostComputed[] = %f --device[] %f\n", i / numCColumns, i % numCColumns, hostComputedC[i], hostC[i]);
            break;
        }
    }

    // printf("\n Number of Blocks Created:%d \n",((numCColumns/Tile_size) + 1)*((numCColumns/Tile_size) + 1));
    // printf("\n Number of Threads Per Block: %d \n",(Tile_size*Tile_size));

    // Free the GPU memory
    funcCheck(hipFree(deviceA));
    funcCheck(hipFree(deviceB));
    funcCheck(hipFree(deviceC));
    funcCheck(hipFree(deviceBias));

    //Free the Pointer Memory
    free(hostA);
    free(hostB);
    free(hostC);
    free(hostBias);
    free(hostComputedC);

    return 0;
}
