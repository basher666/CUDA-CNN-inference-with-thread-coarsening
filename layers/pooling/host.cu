#include "headers.h"
/**
 * Host main routine
 */
int main(void)
{
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float delta = 0.0; //to measure time

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int inp_r=55,  inp_c=55,  depth=96,  filter_width=3,  stride=2,  out_r=27,  out_c=27,  granularity=10;
    int numElements = inp_r*inp_c*depth;
    //Automate calculations of out layer params
    int numElements_out = out_r*out_c*depth;
    size_t size = numElements * sizeof(float);
    size_t size_out = numElements_out * sizeof(float);

    // Allocate the host input vector (will be output of conv layer... won't need to declare this)
    float *h_inp = (float *)malloc(size);
    float *h_out = (float *)malloc(size_out);
    
    // Verify that allocations succeeded
    if (h_inp == NULL || h_out == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_inp[i] = i+1;
    }

    // Allocate the device input vector A
    float *d_inp = NULL;
    err = hipMalloc((void **)&d_inp, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_out = NULL;
    err = hipMalloc((void **)&d_out, size_out);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_inp, h_inp, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the CUDA Kernel
    //Method 1
    printf("\n*****************METHOD 1 (w/o shared mem)*********************\n\n");
    for(granularity=1;granularity<33;granularity++) {
        int threadsPerBlock = 1024;
        int blocksPerGrid =(numElements_out + granularity*threadsPerBlock - 1) / (granularity*threadsPerBlock);
        printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
        hipEventRecord(start);
        gran_pooling<<<blocksPerGrid, threadsPerBlock>>>(d_inp, d_out, inp_r, inp_c, depth, filter_width, stride, out_r, out_c, granularity);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        //delta = 0;
        hipEventElapsedTime(&delta, start, stop);
        printf("granularity = %d, time in milliseconds = %f\n\n", granularity, delta);
	}
	
	
	//Method 2
	printf("\n*****************METHOD 2 (w/ shared mem)*********************\n\n");
	for(granularity=1;granularity<33;granularity++) {
	    int threadsPerBlock = (out_r*out_c - 1)/granularity + 1;
        int blocksPerGrid = depth;
        printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
        hipEventRecord(start);
        shared_pool<<<blocksPerGrid, threadsPerBlock>>>(d_inp, d_out, inp_r, inp_c, depth, filter_width, stride, out_r, out_c, granularity);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        //delta = 0;
        hipEventElapsedTime(&delta, start, stop);
        printf("granularity = %d, time in milliseconds = %f\n\n", granularity, delta);
	}

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_out, d_out, size_out, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements_out; ++i)
    {
        //printf("ELEMENT : %f\n", h_out[i]);
    }

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_inp);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_out);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_inp);
    free(h_out);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}
