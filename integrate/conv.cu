#include "hip/hip_runtime.h"
#include "headers.h"

__device__ float get_ifm_idx(const float* ifm,int k,int i,int j,int in_h,int in_w)
{
  if(i<0 || i>=in_h || j<0 || j>=in_w)
  {
    return 0;
  }
  return ifm[k*in_h*in_w + i*in_w + j];
}
__device__ float get_ifm_plane_idx(float* ifm_plane, int i, int j, int in_h, int in_w)
{
  if(i<0 || i>=in_h || j<0 || j>=in_w)
  {
    return 0;
  }
  return ifm_plane[i*in_w + j];
}

__global__ void conv1_kernel(const float *ifm, float *ofm, float *mask, int in_h, int in_w, int in_n, int out_h, int out_w, int out_m, int mask_size, int pad, int stride, int granularity)
{
  int half_in_h = (in_h + 1)/2;
  int half_in_w = (in_w + 1)/2;
  int out_x = blockIdx.x * blockDim.x + threadIdx.x;
  int out_y = blockIdx.y * half_in_h + threadIdx.y*granularity;
  int out_z = blockIdx.z;
  int in_x = (out_x - pad -1)*stride + mask_size;
  int in_y = (out_y - pad -1)*stride + mask_size;

  if((blockIdx.x == 1 && out_x >= half_in_w) || (blockIdx.y == 1 && out_y >= half_in_h))
  {
    return;
  }


  if(blockIdx.y == 0 && (out_y + granularity) > half_in_h)
  {
    granularity = half_in_h - out_y;
  }
  if(blockIdx.y == 1 && (out_y + granularity) > out_h)
  {
    granularity = out_h - out_y;
  }

  float output[32];  // max granularity = 32
  for(int g=0; g<granularity; g++)
  {
    output[g] = 0.0;
  }

  // computing results
  for(int g=0; g<granularity; g++)
  {
    for(int k=0;k<in_n;k++)
    {
      for(int i=0;i<mask_size;i++)
      {
        for(int j=0;j<mask_size;j++)
        {
          output[g] += ( get_ifm_idx(ifm, k, (in_y + g*stride - i), (in_x - j), in_h, in_w) * mask[out_z*in_n*mask_size*mask_size + k*mask_size*mask_size + i*mask_size + j]);
        }
      }
    }
  }

  for(int g=0;g<granularity;g++)
  {
    ofm[out_z*out_w*out_h + (out_y + g)*out_w + out_x] = 1.0; //(output[g]>0)?output[g]:0;
  }
}

__device__ void load_shared_plane(const float *ifm, float *ifm_plane, int k, int in_n, int in_h, int in_w)
{
  int block_tid = blockDim.x * threadIdx.y + threadIdx.x;
  int total_threads = (blockDim.x * blockDim.y);
  int total_plane_elements = in_h*in_w;
  int elements_per_thread = (total_plane_elements + total_threads - 1)/total_threads;
  int ifm_offset = k*in_n*in_h;
  int curr_ele;

  for(int it = 0; it < elements_per_thread; it++)
  {
    curr_ele = block_tid + (it*total_threads);
    if(curr_ele >= total_plane_elements)
    {
      break;
    }
    ifm_plane[curr_ele] = ifm[ifm_offset + curr_ele];
  }
}

__global__ void conv2_kernel(const float *ifm, float *ofm, float *mask, int in_h, int in_w, int in_n, int out_h, int out_w, int out_m, int mask_size, int pad, int stride, int granularity)  // num threads in a block would be (in_w,ceil(in_h/granularity))
{
  int zflag = 0;

  int out_x = threadIdx.x;
  int out_y = threadIdx.y*granularity;
  int out_z = blockIdx.z;
  int in_x = (out_x - pad -1)*stride + mask_size;
  int in_y = (out_y - pad -1)*stride + mask_size;

  if((out_y + granularity) > out_h)
  {
    granularity = out_h - out_y;
  }

  // float* local_mat = (float *)malloc(granularity*in_n*mask_size*mask_size * sizeof(float));
  // float local_mask[2500];
  __shared__ float shared_mask[2500];
  __shared__ float ifm_plane[3100];

  float output[32];  // max granularity = 32
  for(int g=0; g<granularity; g++)
  {
    output[g] = 0.0;
  }

  // // loading the local mask
  // for(int k=0;k<in_n;k++)
  // {
  //   for(int i=0;i<mask_size;i++)
  //   {
  //     for(int j=0;j<mask_size;j++)
  //     {
  //       local_mask[k*mask_size*mask_size + i*mask_size + j] = mask[out_z*in_n*mask_size*mask_size + k*mask_size*mask_size + i*mask_size +j];
  //     }
  //   }
  // }

  // loading the shared mask
  int total_mask_elements = mask_size*mask_size*in_n;
  int total_threads = (blockDim.x * blockDim.y);
  int elements_per_thread = (total_mask_elements + total_threads - 1)/total_threads;
  int block_tid = blockDim.x * threadIdx.y + threadIdx.x;
  int mask_offset = out_z*total_mask_elements;
  int mask_layer_size = mask_size*mask_size;
  int curr_ele;
  // // printf("blockDim.x = %d, blockDim.y = %d, threadIdx.x = %d, threadIdx.y = %d, total_mask_elements = %d, total_threads = %d, block_tid = %d, mask_offset = %d, mask_layer_size = %d, elements_per_thread = %d\n",blockDim.x,blockDim.y,threadIdx.x, threadIdx.y, total_mask_elements,total_threads,block_tid,mask_offset,mask_layer_size,elements_per_thread);

  for(int it=0; it<elements_per_thread; it++)
  {
    curr_ele = block_tid + (it*total_threads);
    if(curr_ele >= total_mask_elements)
    {
      break;
    }
    shared_mask[curr_ele] = mask[mask_offset + curr_ele];
    // printf("succes for block_tid = %d, it = %d\n",block_tid,it );
    // __syncthreads(); //experiment later
  }
  __syncthreads(); // experiment with the syncthreads in the for loop


  // loading the local ifm
  // for(int g=0; g<granularity && (out_y + g)<out_h; g++)
  // {
  //   for(int k=0; k<in_n; k++)
  //   {
  //     for(int i=0; i<mask_size; i++)
  //     {
  //       for(int j=0; j<mask_size; j++)
  //       {
  //         local_mat[g*total_mask_elements + k*mask_size*mask_size + i*mask_size + j] = 0; //(((in_y + g*stride - i)>=0 && (in_y + g*stride - i) < in_h && (in_x - j)>=0 && (in_x - j) < in_w) ? ifm[k*in_h*in_w + (in_y + g*stride - i)*in_w + (in_x - j)] : 0);
  //       }
  //     }
  //   }
  // }
  
  // //computing output values
  // for(int g=0; g<granularity; g++)
  // {
  //   for(int k=0; k<in_n; k++)
  //   {
  //     for(int i=0; i<mask_size; i++)
  //     {
  //       for(int j=0; j<mask_size; j++)
  //       {
  //         // output[g] += (mask[out_z*in_n*mask_size*mask_size + k*mask_layer_size + i*mask_size + j] * get_ifm_idx(ifm, k, (in_y + g*stride - i), (in_x - j), in_h, in_w));
  //         // output[g] +=  get_ifm_idx(ifm, k, (in_y + g*stride - i), (in_x - j), in_h, in_w) * local_mask[k*mask_size*mask_size + i*mask_size + j];
  //         output[g] +=  get_ifm_idx(ifm, k, (in_y + g*stride - i), (in_x - j), in_h, in_w) * shared_mask[k*mask_size*mask_size + i*mask_size + j];
  //         // if(get_ifm_idx(ifm, k, (in_y + g*stride - i), (in_x - j), in_h, in_w)> eps)
  //         // {
  //         //   zflag = 1;, 
  //         // }
  //       }
  //     }
  //   }
  // }

  //computing output values by accumulating
  for(int k=0; k<in_n; k++)
  {
    load_shared_plane(ifm, ifm_plane, k, in_n, in_h, in_w);
    __syncthreads();
    for(int g=0; g<granularity; g++)
    {
      for(int i=0;i<mask_size;i++)
      {
        for(int j=0;j<mask_size;j++)
        {
          // output[g] +=  get_ifm_plane_idx(ifm_plane, (in_y + g*stride - i), (in_x - j), in_h, in_w) * local_mask[k*mask_size*mask_size + i*mask_size + j];
          output[g] +=  get_ifm_plane_idx(ifm_plane, (in_y + g*stride - i), (in_x - j), in_h, in_w) * shared_mask[k*mask_size*mask_size + i*mask_size + j];
        }
      }
    }
  }
  
  // // printf("out_z = %d, out_w = %d, out_h = %d, out_x = %d, out_y = %d, granularity = %d \n",out_z,out_w,out_h,out_x,out_y,granularity);
  // // writing back to global memory
  for(int g=0;g<granularity;g++)
  {
    ofm[out_z*out_w*out_h + (out_y + g)*out_w + out_x] = (output[g]>0)?output[g]:0;
  }
}

__device__ void load_mask_planes(float *mask, float *mask_planes, int k, int start_layer, int mask_size, int granularity, int in_n, int out_m)
{
  int block_tid = blockDim.x * threadIdx.y + threadIdx.x;
  if(block_tid>=(mask_size*mask_size))
    {
      return;
    }

  int offset = k*mask_size*mask_size;
  int curr_ele, out_idx;
  for(int g=0;g<granularity; g++)
  {
    curr_ele = offset + (start_layer + g)*in_n*mask_size*mask_size + block_tid;
    out_idx = g*mask_size*mask_size + block_tid;
    mask_planes[out_idx] = mask[curr_ele];
  }
}
__global__ void conv3_kernel(const float *ifm, float *ofm, float *mask, int in_h, int in_w, int in_n, int out_h, int out_w, int out_m, int mask_size, int pad, int stride, int granularity)  // num threads in a block would be (in_w,ceil(in_h/granularity))
{
  int zflag = 0;

  int out_x = threadIdx.x;
  int out_y = threadIdx.y;
  int out_z = blockIdx.z*granularity;

  int in_x = (out_x - pad -1)*stride + mask_size;
  int in_y = (out_y - pad -1)*stride + mask_size;

  if((out_z + granularity) > out_m)
  {
    granularity = out_m - out_z;
  }

  __shared__ float mask_planes[500];
  __shared__ float ifm_plane[3030];

  float output[32];  // max granularity = 32
  for(int g=0; g<granularity; g++)
  {
    output[g] = 0.0;
  }

  //computing output values by accumulating
  for(int k=0; k<in_n; k++)
  {
    load_shared_plane(ifm, ifm_plane, k, in_n, in_h, in_w);
    __syncthreads();
    load_mask_planes(mask, mask_planes, k, out_z, mask_size, granularity, in_n, out_m);
    __syncthreads();

    for(int g=0; g<granularity; g++)
    {
      for(int i=0;i<mask_size;i++)
      {
        for(int j=0;j<mask_size;j++)
        {
          output[g] +=  get_ifm_plane_idx(ifm_plane, (in_y - i), (in_x - j), in_h, in_w) * mask_planes[g*mask_size*mask_size + i*mask_size + j];
        }
      }
    }
  }
  
  // // printf("out_z = %d, out_w = %d, out_h = %d, out_x = %d, out_y = %d, granularity = %d \n",out_z,out_w,out_h,out_x,out_y,granularity);
  // // writing back to global memory
  for(int g=0;g<granularity;g++)
  {
    ofm[(out_z + g)*out_w*out_h + out_y*out_w + out_x] = (output[g]>0)?output[g]:0;
  }
}



__global__ void val_checker (float* ifm, float* ofm, float *mask, int ifm_size, int ofm_size, int total_mask_size )
{
  int zflag = 0;
  for(int i=0;i<ifm_size;i++)
  {
    if(fabs(ifm[i])>eps)
    {
      zflag = 1;
    }
  }
  if(zflag==0)
  {
    printf("all ifm elements zero\n");
  }
  __syncthreads();

  zflag = 0;
  for(int i=0;i<ofm_size;i++)
  {
    if(fabs(ofm[i])>eps)
    {
      zflag = 1;
    }
  }
  if(zflag==0)
  {
    printf("all ofm elements zero\n");
  }
  __syncthreads();

  zflag = 0;
  for(int i=0;i<total_mask_size;i++)
  {
    if(fabs(mask[i])>eps)
    {
      zflag = 1;
    }
  }
  if(zflag==0)
  {
    printf("all mask elements zero\n");
  }
  __syncthreads();
}




// __global__ void gen_conv2d(const float *ifm, float *ofm, float *mask, int in_h, int in_w, int in_n,int out_h, int out_w, int out_m, int mask_size, int pad, int stride, int granularity)  // num threads in a block would be (in_w,ceil(in_h/granularity))
// {

//   int out_x = threadIdx.x;
//   int out_y = threadIdx.y;
//   int out_z = blockIdx.z;
//   int in_x = (out_x - pad -1)*stride + mask_size;
//   int in_y = (out_y - pad -1)*stride + mask_size;
//   int mask_radius = mask_size/2;

//   if(out_y + granularity>=out_h)
//   {
//     granularity = out_w - out_y;
//   }

//   __shared__ float shared_mat[granularity][in_n][out_w + mask_size - 1][out_w + mask_size -1];
//   __shared__ float shared_mask[in_n][mask_size][mask_size];
//   float output[granularity];
//   for(int g=0;g<granularity;g)
//   {
//     output[g] = 0.0;
//   }

//   for(int g=0;g<granularity && (out_y + g)<out_h ;g++)
//   {
//     for(int k=0;k<in_n;k++)
//     {
//       // loading left halo elements
//       if(out_x<)
//       shared_mat[g][k][out_y + mask_radius][out_x] = 

//       // loading center element
//       shared_mat[g][k][out_y + mask_radius][out_x + mask_radius] = (((in_y - out_y)>=0 && (in_y - out_y)<in_h && (in_x - out_x)>=0 && (in_x - out_x)<in_w) ? ifm[k*in_h*in_w + (in_y + g - out_y)*in_w + (in_x - out_x)] : 0);
//       // __syncthreads();  // maybe ?: experiment for better speedup
//     }
//   }
  
//   __syncthreads(); // maybe ?

//   if(out_x<mask_size && out_y<mask_size)
//   {
//     for(int k=0;k<in_n;k++)
//     {
//       local_mask[k][out_y][out_x] = mask[k*mask_size*mask_size + out_y*mask_size + out_x];

//       // __syncthreads(); //experiment later 
//     }
//   }

//   __syncthreads(); // experiment with the syncthreads in the for loop

//   for(int g=0;g<granularity;g++)
//   {
//     for(int k=0;k<in_n;k++)
//     {
//       for(int i=0;i<mask_size;i++)
//       {
//         for(int j=0;j<mask_size;j++)
//         {
//           output[g] += local_mask[k][i][j] * local_mat[g][k][out_y][out_x];
//         }

//       }
//     }
//   }

//   for(int g=0;g<granularity;g++)
//   {
//     ofm[out_z*out_w*out_w + (out_y + g)*out_w + out_x] = output[g];
//   }
// }


