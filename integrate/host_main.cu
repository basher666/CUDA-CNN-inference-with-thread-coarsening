#include <vector>
#include <iostream>
#include <fstream>
#include <locale>
#include <limits>
#include <sstream>
#include <algorithm>
#include "headers.h"


int main(){

	hipEvent_t start, stop;
	hipEventCreate(&start);
	float delta = 0.0; //to measure time
	hipError_t err = hipSuccess;
	std::string line;
	std::ifstream in("alexnet.csv");

	float *d_ofm, *d_out, *d_ofm_3, *d_ofm_4, *d_ofm_5, *out_1, *out_2, *out_3;
	ConvLayer conv_1, conv2, conv3, conv4, conv5;
	FCLayer fc6 , fc7, fc8;
	int granularity = 1;
	printf(" start \n");
	while(getline(in, line)) {
		std::stringstream lineStream(line);
    	if (line[0] == 'c') { //Convolutional layer 
    		ConvLayer c = processConv(lineStream);
    		printf("%s c.name %d \n",c.name.c_str(),c.filter_size);
    		if(c.name == "conv1"){ conv_1 = c;}
    		if(c.name == "conv2"){ conv2 = c;}
    		if(c.name == "conv3"){ conv3 = c;}
    		if(c.name == "conv4"){ conv4 = c;}
    		if(c.name == "conv5"){ conv5 = c;}
    	} else if (line[0] == 'f') { //Fully Connected Layer
    		FCLayer f = processFC(lineStream);
    		if(f.name == "fc6"){ fc6 = f;}
    		if(f.name == "fc7"){ fc7 = f;}
    		if(f.name == "fc8"){ fc8 = f;}
    	}
    }
    printf("load complete \n");
	for(granularity =1; granularity <= 16; granularity++) {
		// Convolution layer 1
		if(true){
	
			// i/p : 3x227x227, o/p : 96x55x55, filter : 11x11x96x3
			int num_in_fm = 3;
		   	int in_fm_h = 227;
		   	int in_fm_w = 227;
		   	int num_out_fm = 96;
		   	int out_fm_w = 55;
		   	int out_fm_h = 55;
		   	int mask_size = 11;
		   	int stride = 4;
		   	int pad = 0;
		   	int in_size = num_in_fm*in_fm_w*in_fm_h * sizeof(float);
		   	printf("57\n");
		   	float *h_ifm = (float*) malloc(in_size);
		   	printf("59\n");
		   	// random generation of the i/p image matrix 
		   	for(int i=0;i< num_in_fm*in_fm_w*in_fm_h;i++){
		   		h_ifm[i] = rand()/(float) RAND_MAX;
		   	}
		   	int out_size = num_out_fm*out_fm_w*out_fm_w * sizeof(float);
		   	int total_mask_size = num_out_fm*num_in_fm*mask_size*mask_size*sizeof(float);
		   	printf(" In conv 1 \n");
			printf(" filter_size : %d , num_layers : %d, depth : %d \n",conv_1.filter_size,conv_1.num_layers,conv_1.depth);
	
		   	float *d_ifm = NULL;
		    err = hipMalloc((void **)&d_ifm, in_size);
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to allocate device ifm (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
						printf("78\n");

		    d_ofm = NULL;
		    err = hipMalloc((void **)&d_ofm, out_size);
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to allocate device ofm (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
						printf("87\n");

		    float *d_mask = NULL;
		    err = hipMalloc((void **)&d_mask, total_mask_size);
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to allocate device mask (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
				printf("96\n");

		    err = hipMemcpy(d_ifm, h_ifm, in_size, hipMemcpyHostToDevice);
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to copy matrix ifm from host to device (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
			printf("101\n");
			float *h_mask = conv_1.weights;
			err = hipMemcpy(d_mask, h_mask, total_mask_size, hipMemcpyHostToDevice);
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to copy matrix mask from host to device (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
				printf("108\n");

		
		    dim3 blocksPerGrid(num_out_fm,1,1);
		    dim3 threadsPerBlock(out_fm_w, ((out_fm_h + granularity - 1)/granularity) , 1);
			printf("threadsPerBlock for Conv1 = %d,%d,%d\n",threadsPerBlock.x,threadsPerBlock.y,threadsPerBlock.z);
	
			hipEventRecord(start);
	
		    conv1_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_ifm, d_ofm, d_mask, in_fm_h, in_fm_w, num_in_fm, out_fm_h, out_fm_w, num_out_fm, mask_size, pad, stride, granularity);
	    	
		    // d_ofm will now be used for the further layers 
		    err = hipFree(d_ifm);
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to free device matrix ifm (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
		    err = hipFree(d_mask);
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to free device matrix mask (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
	
		    // Free host memory
		    free(h_ifm);
		    free(h_mask);
	
		}
		printf("conv1 done \n");
		// maxpooling 1 
		{
			// i/p : 96x55x55 , filter : 3x3 , stride 2 , o/p : 96x27x27
			int inp_r=55,  inp_c=55,  depth=96,  filter_width=3,  stride=2,  out_r=27,  out_c=27;
		    int numElements = inp_r*inp_c*depth;
		    int numElements_out = out_r*out_c*depth;
		    size_t size = numElements * sizeof(float);
		    size_t size_out = numElements_out * sizeof(float);
	
		    // Allocate the device output vector C
		    d_out = NULL;
		    err = hipMalloc((void **)&d_out, size_out);
	
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
	
		    
        	 dim3 blocksPerGrid(depth,1,1);
		    dim3 threadsPerBlock((out_r*out_c - 1)/granularity + 1,1 , 1);
		    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid.x, threadsPerBlock.x);
	
		    // d_ofm is the o/p from the layer, will be the i/p of this 
		    shared_pool<<<blocksPerGrid, threadsPerBlock>>>(d_ofm, d_out, inp_r, inp_c, depth, filter_width, stride, out_r, out_c, granularity);
		    err = hipGetLastError();
	
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
	
		    // Free device global memory , d_ofm is the o/p of the Conv1 , not needed amymore 
		    err = hipFree(d_ofm);
	
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
	
		} // o/p of maxpooling is in d_out 
		printf("maxpooling 1 Done. \n");
		// Conv2
		{
			if(true){
	
				// i/p : 96x27x27, o/p : 256x27x27, filter : 5x5x256x48 , padding : 2 
				int num_in_fm = 48;
			   	int in_fm_h = 27;
			   	int in_fm_w = 27;
			   	int num_out_fm = 256;
			   	int out_fm_w = 27;
			   	int out_fm_h = 27;
			   	int mask_size = 5;
			   	int stride = 1;
			   	int pad = 2;
			   	int in_size = num_in_fm*in_fm_w*in_fm_h * sizeof(float);
	
			   	int out_size = num_out_fm*out_fm_w*out_fm_w * sizeof(float);
			   	int total_mask_size = num_out_fm*num_in_fm*mask_size*mask_size*sizeof(float);
			   	float *h_mask = conv2.weights;
			   	printf(" In conv 2 \n");
				printf(" filter_size : %d , num_layers : %d, depth : %d \n",conv2.filter_size,conv2.num_layers,conv2.depth);
	
			   	
			    d_ofm = NULL;
			    err = hipMalloc((void **)&d_ofm, out_size);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to allocate device ofm (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
			    float *d_mask = NULL;
			    err = hipMalloc((void **)&d_mask, total_mask_size);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to allocate device mask (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
				err = hipMemcpy(d_mask, h_mask, total_mask_size, hipMemcpyHostToDevice);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to copy matrix mask from host to device (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
			
			    dim3 blocksPerGrid(num_out_fm,1,1);
			    dim3 threadsPerBlock(out_fm_w, ((out_fm_h + granularity - 1)/granularity) , 1);
				printf("threadsPerBlock for Conv1 = %d,%d,%d\n",threadsPerBlock.x,threadsPerBlock.y,threadsPerBlock.z);
	
				hipEventRecord(start);
	
			    conv2_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_out, d_ofm, d_mask, in_fm_h, in_fm_w, num_in_fm, out_fm_h, out_fm_w, num_out_fm, mask_size, pad, stride, granularity);
		    	
			    // d_ofm will now be used for the further layers 
			    err = hipFree(d_out);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to free device matrix ifm (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
			    err = hipFree(d_mask);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to free device matrix mask (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
			    // Free host memory
			    free(h_mask);
	
			} 
		}	// o/p is d_ofm 
		printf("Conv2 Done \n");
		// maxpooling 2
		{
			// i/p : 256x27x27 , filter : 3x3 , stride 2 , o/p : 265x13x13
			int inp_r=27,  inp_c=27,  depth=256,  filter_width=3,  stride=2,  out_r=13,  out_c=13;
		    int numElements = inp_r*inp_c*depth;
		    int numElements_out = out_r*out_c*depth;
		    size_t size = numElements * sizeof(float);
		    size_t size_out = numElements_out * sizeof(float);
	
		    // Allocate the device output vector C
		    d_out = NULL;
		    err = hipMalloc((void **)&d_out, size_out);
	
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
	
	     	int threadsPerBlock = (out_r*out_c - 1)/granularity + 1;
        	int blocksPerGrid = depth;
		    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	
		    // d_ofm is the o/p from the layer, will be the i/p of this 
		    shared_pool<<<blocksPerGrid, threadsPerBlock>>>(d_ofm, d_out, inp_r, inp_c, depth, filter_width, stride, out_r, out_c, granularity);
		    err = hipGetLastError();
	
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
	
		    // Free device global memory , d_ofm is the o/p of the Conv1 , not needed amymore 
		    err = hipFree(d_ofm);
	
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
	
		} // o/p of maxpooling is in d_out 
		printf("maxpooling 2 \n");
		// Conv3
		{
			 
			if(true){
	
				// i/p : 256x13x13, o/p : 384x13x13, filter : 3x3x384x256 , padding : 1 
				int num_in_fm = 256;
			   	int in_fm_h = 13;
			   	int in_fm_w = 13;
			   	int num_out_fm = 384;
			   	int out_fm_w = 13;
			   	int out_fm_h = 13;
			   	int mask_size = 3;
			   	int stride = 1;
			   	int pad = 1;
			   	int in_size = num_in_fm*in_fm_w*in_fm_h * sizeof(float);
	
			   	int out_size = num_out_fm*out_fm_w*out_fm_w * sizeof(float);
			   	int total_mask_size = num_out_fm*num_in_fm*mask_size*mask_size*sizeof(float);
			   	float *h_mask = conv3.weights;
				printf(" In conv 3 \n");
				printf(" filter_size : %d , num_layers : %d, depth : %d \n",conv3.filter_size,conv3.num_layers,conv3.depth);
	
			    d_ofm_3 = NULL;
			    err = hipMalloc((void **)&d_ofm_3, out_size);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to allocate device ofm (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
			    float *d_mask = NULL;
			    err = hipMalloc((void **)&d_mask, total_mask_size);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to allocate device mask (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
				err = hipMemcpy(d_mask, h_mask, total_mask_size, hipMemcpyHostToDevice);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to copy matrix mask from host to device (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
			
			    dim3 blocksPerGrid(num_out_fm,1,1);
			    dim3 threadsPerBlock(out_fm_w, ((out_fm_h + granularity - 1)/granularity) , 1);
				printf("threadsPerBlock for Conv1 = %d,%d,%d\n",threadsPerBlock.x,threadsPerBlock.y,threadsPerBlock.z);
	
				hipEventRecord(start);
	
			    conv2_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_out, d_ofm_3, d_mask, in_fm_h, in_fm_w, num_in_fm, out_fm_h, out_fm_w, num_out_fm, mask_size, pad, stride, granularity);
		    	
			    // d_ofm will now be used for the further layers 
			    err = hipFree(d_out);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to free device matrix ifm (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
			    err = hipFree(d_mask);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to free device matrix mask (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
			    // Free host memory
			    free(h_mask);
	
			} 
		}	// o/p is d_ofm_3 
		printf("Conv3 Done\n");
		// Conv4
		{
			 
	
			if(true){
				ConvLayer c = conv4;
	
				// i/p : 384x13x13, o/p : 384x13x13, filter : 3x3x384x192 , padding : 1 
				int num_in_fm = 192;
			   	int in_fm_h = 13;
			   	int in_fm_w = 13;
			   	int num_out_fm = 384;
			   	int out_fm_w = 13;
			   	int out_fm_h = 13;
			   	int mask_size = 3;
			   	int stride = 1;
			   	int pad = 2;
			   	int in_size = num_in_fm*in_fm_w*in_fm_h * sizeof(float);
	
			   	int out_size = num_out_fm*out_fm_w*out_fm_w * sizeof(float);
			   	int total_mask_size = num_out_fm*num_in_fm*mask_size*mask_size*sizeof(float);
			   	float *h_mask = c.weights;
				printf(" In conv 4 \n");
				printf(" filter_size : %d , num_layers : %d, depth : %d \n",conv4.filter_size,conv4.num_layers,conv4.depth);
		
			    d_ofm_4 = NULL;
			    err = hipMalloc((void **)&d_ofm_4, out_size);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to allocate device ofm (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
			    float *d_mask = NULL;
			    err = hipMalloc((void **)&d_mask, total_mask_size);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to allocate device mask (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
				err = hipMemcpy(d_mask, h_mask, total_mask_size, hipMemcpyHostToDevice);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to copy matrix mask from host to device (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
			
			    dim3 blocksPerGrid(num_out_fm,1,1);
			    dim3 threadsPerBlock(out_fm_w, ((out_fm_h + granularity - 1)/granularity) , 1);
				printf("threadsPerBlock for Conv1 = %d,%d,%d\n",threadsPerBlock.x,threadsPerBlock.y,threadsPerBlock.z);
	
				hipEventRecord(start);
	
			    conv2_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_ofm_3, d_ofm_4, d_mask, in_fm_h, in_fm_w, num_in_fm, out_fm_h, out_fm_w, num_out_fm, mask_size, pad, stride, granularity);
		    	
			    // d_ofm will now be used for the further layers 
			    err = hipFree(d_ofm_3);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to free device matrix ifm (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
			    err = hipFree(d_mask);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to free device matrix mask (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
			    // Free host memory
			    free(h_mask);
	
			} 
		}	// o/p is d_ofm_4 
		printf("Conv4 Done\n");
		// Conv5
		{
			 
			getline(in, line);
			std::stringstream lineStream(line);
			if(line[0] == 'c'){
				ConvLayer c = processConv(lineStream);
	
				// i/p : 384x13x13, o/p : 256x13x13, filter : 3x3x256x192 , padding : 1 
				int num_in_fm = 192;
			   	int in_fm_h = 13;
			   	int in_fm_w = 13;
			   	int num_out_fm = 256;
			   	int out_fm_w = 13;
			   	int out_fm_h = 13;
			   	int mask_size = 3;
			   	int stride = 1;
			   	int pad = 2;
			   	int in_size = num_in_fm*in_fm_w*in_fm_h * sizeof(float);
	
			   	int out_size = num_out_fm*out_fm_w*out_fm_w * sizeof(float);
			   	int total_mask_size = num_out_fm*num_in_fm*mask_size*mask_size*sizeof(float);
			   	float *h_mask = conv5.weights;
			   	float *test_ofm = (float *) malloc(out_size);
				printf(" In conv 5 \n");
				printf(" filter_size : %d , num_layers : %d, depth : %d \n",conv5.filter_size,conv5.num_layers,conv5.depth);
	
			    d_ofm_5 = NULL;
			    err = hipMalloc((void **)&d_ofm_5, out_size);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to allocate device ofm (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
			    float *d_mask = NULL;
			    err = hipMalloc((void **)&d_mask, total_mask_size);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to allocate device mask (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
				err = hipMemcpy(d_mask, h_mask, total_mask_size, hipMemcpyHostToDevice);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to copy matrix mask from host to device (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
			
			    dim3 blocksPerGrid(num_out_fm,1,1);
			    dim3 threadsPerBlock(out_fm_w, ((out_fm_h + granularity - 1)/granularity) , 1);
				printf("threadsPerBlock for Conv1 = %d,%d,%d\n",threadsPerBlock.x,threadsPerBlock.y,threadsPerBlock.z);
	
				hipEventRecord(start);
	
			    conv2_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_ofm_4, d_ofm_5, d_mask, in_fm_h, in_fm_w, num_in_fm, out_fm_h, out_fm_w, num_out_fm, mask_size, pad, stride, granularity);
		    	
			    // d_ofm will now be used for the further layers 
			    err = hipFree(d_ofm_4);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to free device matrix ifm (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
			    err = hipFree(d_mask);
			    if (err != hipSuccess)
			    {
			        fprintf(stderr, "Failed to free device matrix mask (error code %s)!\n", hipGetErrorString(err));
			        exit(EXIT_FAILURE);
			    }
	
			    // Free host memory
			    free(h_mask);
	
			} 
		}	// o/p id d_ofm_5
		printf("Conv5 Done \n");
		// maxpooling 3
		{
			// i/p : 256x13x13 , filter : 3x3 , stride 2 , o/p : 265x6x6
			int inp_r=13,  inp_c=13,  depth=256,  filter_width=3,  stride=2,  out_r=6,  out_c=6;
		    int numElements = inp_r*inp_c*depth;
		    int numElements_out = out_r*out_c*depth;
		    size_t size = numElements * sizeof(float);
		    size_t size_out = numElements_out * sizeof(float);
	
		    // Allocate the device output vector C
		    d_out = NULL;
		    err = hipMalloc((void **)&d_out, size_out);
	
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
	
     		int threadsPerBlock = (out_r*out_c - 1)/granularity + 1;
        	int blocksPerGrid = depth;
		    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	
		    // d_ofm is the o/p from the layer, will be the i/p of this 
		    shared_pool<<<blocksPerGrid, threadsPerBlock>>>(d_ofm_5, d_out, inp_r, inp_c, depth, filter_width, stride, out_r, out_c, granularity);
		    err = hipGetLastError();
	
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
	
		    // Free device global memory , d_ofm is the o/p of the Conv1 , not needed amymore 
		    err = hipFree(d_ofm);
	
		    if (err != hipSuccess)
		    {
		        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
		        exit(EXIT_FAILURE);
		    }
	
		} // o/p of maxpooling is in d_out 
		printf("maxpooling 3 done \n");
		// FC 6 
		{
			// if_vector : 256x6x6 matrix : 
			FCLayer f = fc6;
	    	int numARows = fc6.outputs;   // number of rows in the ifm 
			int numAColumns = fc6.inputs;  // number of columns in the ifm
			int numBRows = 256*6*6;   // number of rows in the vector
			int numBColumns=1;  // number of columns in the vector
			int numCRows = numARows;  // number of rows in the matrix C (you have to set this)
			int numCColumns=1; // number of columns in the matrix C (you have to set this)
			int nelem_per_thread = granularity; // THread coarsening factor
			float  *hostBias, *deviceBias, *matrix;
			hipError_t err = hipMalloc((void **)&out_1, sizeof(float)*numCRows*numCColumns);
			if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	        hostBias = fc6.biases;
	        err = hipMalloc((void **)&deviceBias, sizeof(float)*numCRows*numCColumns);
	        if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	
	        err = hipMemcpy(deviceBias, hostBias, sizeof(float)*numCRows*numCColumns, hipMemcpyHostToDevice);
	        if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }

	        err = hipMalloc((void **)&matrix, sizeof(float)*fc6.outputs*fc6.inputs);
			if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	
	        err = hipMemcpy(matrix, fc6.weights, sizeof(float)*f.outputs*f.inputs, hipMemcpyHostToDevice);
	        if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	
	        // Initialize the grid and block dimensions
		    // Launch the Vector Add CUDA Kernel
		    int numThreadsReq = (numCRows+nelem_per_thread-1)/nelem_per_thread;
		    int threadsPerBlock = 256;
		    int blocksPerGrid =(numThreadsReq + threadsPerBlock - 1) / threadsPerBlock;
		    dim3 dimGrid(blocksPerGrid, 1, 1);//Number of Blocks required
		    dim3 dimBlock(threadsPerBlock, 1, 1);//Number of threads in each block
			
		    // Shared memory for parameter vetor and bias values
		    int totSharedMem = (numAColumns + numCRows*numCColumns)* sizeof(float); // Shared memory per block
		    printf("CUDA kernel launch with %d blocks of %d threads, and %d of shared Memory\n", blocksPerGrid, threadsPerBlock, totSharedMem);
	
		    gen_matvec<<<dimGrid, dimBlock, totSharedMem>>>(matrix, d_out, out_1, deviceBias, numCRows, numAColumns, nelem_per_thread);
	
		    hipError_t err1 = hipPeekAtLastError();//To capture last error in function call
	
		    err = hipFree(matrix);
			if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	
	        err = hipFree(d_out);
	        if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	
		}	 // out is the output 
		printf("FC6 done\n");
	
		// FC7 
		{
			// ip matrix : 4096x4096 , output vector : 4096x1 
			FCLayer f = fc7;
	    	int numARows = f.outputs;   // number of rows in the ifm 
			int numAColumns = f.inputs;  // number of columns in the ifm
			int numBRows = 4096;   // number of rows in the ofm
			int numBColumns=1;  // number of columns in the ofm
			int numCRows = numARows;  // number of rows in the matrix C (you have to set this)
			int numCColumns=1; // number of columns in the matrix C (you have to set this)
			int nelem_per_thread = granularity; // THread coarsening factor
			float  *hostBias, *deviceBias, *matrix;
			hipError_t err = hipMalloc((void **)&out_2, sizeof(float)*numCRows*numCColumns);
			if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	        hostBias = f.biases;
	        err = hipMalloc((void **)&deviceBias, sizeof(float)*numCRows*numCColumns);
	        if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	
	        err = hipMemcpy(deviceBias, hostBias, sizeof(float)*numCRows*numCColumns, hipMemcpyHostToDevice);
	        if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }

	        err = hipMalloc((void **)&matrix, sizeof(float)*f.outputs*f.inputs);
			if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	
	        err = hipMemcpy(matrix, f.weights, sizeof(float)*f.outputs*f.inputs, hipMemcpyHostToDevice);
	        if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	
	        // Initialize the grid and block dimensions
		    // Launch the Vector Add CUDA Kernel
		    int numThreadsReq = (numCRows+nelem_per_thread-1)/nelem_per_thread;
		    int threadsPerBlock = 256;
		    int blocksPerGrid =(numThreadsReq + threadsPerBlock - 1) / threadsPerBlock;
		    dim3 dimGrid(blocksPerGrid, 1, 1);//Number of Blocks required
		    dim3 dimBlock(threadsPerBlock, 1, 1);//Number of threads in each block
			
		    // Shared memory for parameter vetor and bias values
		    int totSharedMem = (numAColumns + numCRows*numCColumns)* sizeof(float); // Shared memory per block
		    printf("CUDA kernel launch with %d blocks of %d threads, and %d of shared Memory\n", blocksPerGrid, threadsPerBlock, totSharedMem);
	
		    gen_matvec<<<dimGrid, dimBlock, totSharedMem>>>(matrix, out_1, out_2, deviceBias, numCRows, numAColumns, nelem_per_thread);
	
		    hipError_t err1 = hipPeekAtLastError();//To capture last error in function call
	
		    err = hipFree(matrix);
			if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	
	        err = hipFree(out_1);
	        if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	
		}	 // out_2 is the output 
		printf("FC7 done \n");
		// FC8
		{
			// ip matrix : 1000x4096 , output vector : 1000x1 
			FCLayer f = fc8;
	    	int numARows = f.outputs;   // number of rows in the ifm 
			int numAColumns = f.inputs;  // number of columns in the ifm
			int numBRows = 4096;   // number of rows in the vector
			int numBColumns=1;  // number of columns in the vector
			int numCRows = numARows;  // number of rows in the matrix C (you have to set this)
			int numCColumns=1; // number of columns in the matrix C (you have to set this)
			int nelem_per_thread = granularity; // THread coarsening factor
			float  *hostBias, *deviceBias, *matrix;
			hipError_t err = hipMalloc((void **)&out_3, sizeof(float)*numCRows*numCColumns);
			if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	        hostBias = f.biases;
	        err = hipMalloc((void **)&deviceBias, sizeof(float)*numCRows*numCColumns);
	        if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	
	        err = hipMemcpy(deviceBias, hostBias, sizeof(float)*numCRows*numCColumns, hipMemcpyHostToDevice);
	        if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	

			err = hipMalloc((void **)&matrix, sizeof(float)*f.outputs*f.inputs);
			if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }

	        err = hipMemcpy(matrix, f.weights, sizeof(float)*f.outputs*f.inputs, hipMemcpyHostToDevice);
	        if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	
	        // Initialize the grid and block dimensions
		    // Launch the Vector Add CUDA Kernel
		    int numThreadsReq = (numCRows+nelem_per_thread-1)/nelem_per_thread;
		    int threadsPerBlock = 256;
		    int blocksPerGrid =(numThreadsReq + threadsPerBlock - 1) / threadsPerBlock;
		    dim3 dimGrid(blocksPerGrid, 1, 1);//Number of Blocks required
		    dim3 dimBlock(threadsPerBlock, 1, 1);//Number of threads in each block
			
		    // Shared memory for parameter vetor and bias values
		    int totSharedMem = (numAColumns + numCRows*numCColumns)* sizeof(float); // Shared memory per block
		    printf("CUDA kernel launch with %d blocks of %d threads, and %d of shared Memory\n", blocksPerGrid, threadsPerBlock, totSharedMem);
	
		    gen_matvec<<<dimGrid, dimBlock, totSharedMem>>>(matrix, out_2, out_3, deviceBias, numCRows, numAColumns, nelem_per_thread);
	
		    hipError_t err1 = hipPeekAtLastError();//To capture last error in function call
	
		    err = hipFree(matrix);
			if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	
	        err = hipFree(out_2);
	        if (err != hipSuccess) {
	            printf( "Failed to run stmt %d ", __LINE__);
	            return -1;
	        }
	
		}	 // out_2 is the output 

		printf("FC8 Done \n");
		hipEventRecord(stop);
	    hipEventSynchronize(stop);
		delta = 0;
		hipEventElapsedTime(&delta, start, stop);
		printf("conv2, shared_pool, gen_matvec\n");
		printf("granularity = %d, time in milliseconds = %f\n",granularity,delta);
	}



	
    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;


}